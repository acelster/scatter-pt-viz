#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <omp.h>

#include "node.h"
#include "point.h"
#include "color.h"
#include "global.h"
#include "settings.h"
#include "real.h"
#include "raycreator.h"

#define NODE_CACHE_SIZE 128
//#define TEXTURE

//Preallocating for up to 4 devices, a little wasteful if fewer are used...
Node* nodes_d[4];
Point* points_d[4];
Display_color* image_d[4];
Ray* rays_d[4];
Color* colors_d[4];
unsigned int * stack_d[4];

__device__ __constant__ real_t STEP_SIZE_D;
__device__ __constant__ real_t INTERPOLATION_RADIUS_D;
__device__ __constant__ real_t X_RADIUS;
__device__ __constant__ real_t Y_RADIUS;
__device__ __constant__ real_t Z_RADIUS;
__device__ __constant__ real_t STEP_FACTOR_D;
__device__ __constant__ real_t STEP_LIMIT_D;
__device__ __constant__ real_t MAX_INTENSITY_D;
__device__ __constant__ Ranges top_ranges;
__device__ __constant__ Node root_node;
__device__ __constant__ real_t aniso_x;
__device__ __constant__ real_t aniso_y;
__device__ __constant__ real_t aniso_z;

//From raycreator
__device__ __constant__ Coord up;
__device__ __constant__ Coord eye;
__device__ __constant__ Coord screen_center_d;
__device__ __constant__ Coord right;
__device__ __constant__ Coord top_d;
__device__ __constant__ Coord bottom_d;

__device__ __constant__ int RESOLUTION_D;

#ifdef TEXTURE
texture<float4, hipTextureType1D, hipReadModeElementType> pointTexture;
#endif

inline __device__ Coord make_Coord(real_t x, real_t y, real_t z){
  Coord c = {x,y,z};
  return c;
}

inline __device__ Coord operator+(Coord a, Coord b){
  return make_Coord(a.x + b.x, a.y + b.y, a.z + b.z);
}

inline __device__ Coord operator-(Coord a, Coord b){
  return make_Coord(a.x - b.x, a.y - b.y, a.z - b.z);
}

inline __device__ Coord operator*(Coord a, real_t s){
  return make_Coord(a.x * s, a.y * s, a.z * s);
}

inline __device__ Coord operator/(Coord a, Coord b){
  return make_Coord(a.x / b.x, a.y / b.y, a.z / b.z);
}

static __inline__ __device__ Coord fminf(Coord a, Coord b){
  return make_Coord(fminf(a.x,b.x), fminf(a.y,b.y), fminf(a.z,b.z));
}

static __inline__ __device__ Coord fmaxf(Coord a, Coord b){
  return make_Coord(fmaxf(a.x,b.x), fmaxf(a.y,b.y), fmaxf(a.z,b.z));
}

__device__ Ray normalize_ray(Ray r){
  Ray t = r;
  real_t d = sqrt(r.dir.x*r.dir.x + r.dir.y*r.dir.y + r.dir.z*r.dir.z);
  t.dir.x = t.dir.x / d;
  t.dir.y = t.dir.y / d;
  t.dir.z = t.dir.z / d;

  return t;
}

__device__ Color blend_d(Color c, real_t intensity, Color* colors){
  Color new_color, output;

  new_color = colors[(int)((intensity/MAX_INTENSITY_D)*10000)];

  output.r = c.r + (1-c.a)*new_color.a*new_color.r;
  output.g = c.g + (1-c.a)*new_color.a*new_color.g;
  output.b = c.b + (1-c.a)*new_color.a*new_color.b;

  output.a = c.a + (1-c.a)*new_color.a;

  return output;
}

__device__ Display_color to_display_color_d(Color c){
  Display_color dc;

  if(c.r > 1.0f){
    c.r = 1.0;
  }

  if(c.b > 1.0f){
    c.b = 1.0;
  }

  if(c.g > 1.0f){
    c.g = 1.0;
  }
  
  dc.a = (unsigned char)(c.a*255);
  dc.r = (unsigned char)(c.r*255);
  dc.g = (unsigned char)(c.g*255);
  dc.b = (unsigned char)(c.b*255);

  return dc;
}

__device__ int get_index_d(Coord pos, Ranges r){
  char a = pos.x < r.xmin + (r.xmax - r.xmin)/2.0f;
  char b = pos.y < r.ymin + (r.ymax - r.ymin)/2.0f;
  char c = pos.z < r.zmin + (r.zmax - r.zmin)/2.0f;

  return a*4 + b*2 + c*1;
}

__device__ Ranges get_ranges_for_index_d(Ranges r, int index){
  Ranges nr = r;
  if(index == 0){
    nr.xmin += (r.xmax -r.xmin)/2.0f;
    nr.ymin += (r.ymax -r.ymin)/2.0f;
    nr.zmin += (r.zmax -r.zmin)/2.0f;
  }
  if(index == 1){
    nr.xmin += (r.xmax -r.xmin)/2.0f;
    nr.ymin += (r.ymax -r.ymin)/2.0f;
    nr.zmax -= (r.zmax -r.zmin)/2.0f;
  }
  if(index == 2){
    nr.xmin += (r.xmax -r.xmin)/2.0f;
    nr.ymax -= (r.ymax -r.ymin)/2.0f;
    nr.zmin += (r.zmax -r.zmin)/2.0f;
  }
  if(index == 3){
    nr.xmin += (r.xmax -r.xmin)/2.0f;
    nr.ymax -= (r.ymax -r.ymin)/2.0f;
    nr.zmax -= (r.zmax -r.zmin)/2.0f;
  }
  if(index == 4){
    nr.xmax -= (r.xmax -r.xmin)/2.0f;
    nr.ymin += (r.ymax -r.ymin)/2.0f;
    nr.zmin += (r.zmax -r.zmin)/2.0f;
  }
  if(index == 5){
    nr.xmax -= (r.xmax -r.xmin)/2.0f;
    nr.ymin += (r.ymax -r.ymin)/2.0f;
    nr.zmax -= (r.zmax -r.zmin)/2.0f;
  }
  if(index == 6){
    nr.xmax -= (r.xmax -r.xmin)/2.0f;
    nr.ymax -= (r.ymax -r.ymin)/2.0f;
    nr.zmin += (r.zmax -r.zmin)/2.0f;
  }
  if(index == 7){
    nr.xmax -= (r.xmax -r.xmin)/2.0f;
    nr.ymax -= (r.ymax -r.ymin)/2.0f;
    nr.zmax -= (r.zmax -r.zmin)/2.0f;
  }

  return nr;
}

__device__ real_t interpolate(Node n, Coord pos, Point* points){
  real_t weight = 0; 
  real_t intensity = 0;

  for(int c = 0; c < n.num_children; c++){
    real_t dx = pos.x - points[(long int)n.pointer + c].x;
    real_t dy = pos.y - points[(long int)n.pointer + c].y;
    real_t dz = pos.z - points[(long int)n.pointer + c].z;

    real_t distance = sqrt(dx*dx + dy*dy + dz*dz);

    if(distance < INTERPOLATION_RADIUS_D){
      intensity += (1/distance)*points[(long int)n.pointer + c].intensity;
      weight += (1/distance);
    }
  }

  if(intensity <= 0){
    return 0;
  }

  real_t ratio = intensity/weight;
  if(ratio <= 1){
    return 0;
  }

  return log(ratio);
}

inline __device__ unsigned int get_covered_subnodes(Coord pos, Node n){

//    real_t X_RADIUS = INTERPOLATION_RADIUS_D;
 //   real_t Y_RADIUS = INTERPOLATION_RADIUS_D;
  //  real_t Z_RADIUS = INTERPOLATION_RADIUS_D;

  char xmin = pos.x - X_RADIUS < n.x;
  char ymin = pos.y - Y_RADIUS < n.y;
  char zmin = pos.z - Z_RADIUS < n.z;

  char xmax = pos.x + X_RADIUS > n.x;
  char ymax = pos.y + Y_RADIUS > n.y;
  char zmax = pos.z + Z_RADIUS > n.z;

  unsigned char b = 0;

  b = b | (unsigned int)(128 * (xmin && ymin && zmin));
  b = b | (unsigned int)(64 * (xmin && ymin && zmax));
  b = b | (unsigned int)(32 * (xmin && ymax && zmin));
  b = b | (unsigned int)(16 * (xmin && ymax && zmax));
  b = b | (unsigned int)(8 * (xmax && ymin && zmin));
  b = b | (unsigned int)(4 * (xmax && ymin && zmax));
  b = b | (unsigned int)(2 * (xmax && ymax && zmin));
  b = b | (unsigned int)(1 * (xmax && ymax && zmax));

  return b;
}

#ifdef TEXTURE
__device__ real_t get_intensity_for_pos_full(Coord pos, Node* nodes, Node* node_cache, unsigned int* stack, int base){
#else
__device__ real_t get_intensity_for_pos_full(Coord pos, const __restrict__ Point* points, Node* nodes, Node* node_cache, unsigned int* stack, int base){
#endif

  real_t intensity = 0;
  real_t weight = 0;
  short tos = 0;
  unsigned char inc = 1;

  unsigned int current_node = 0;
  unsigned int b = get_covered_subnodes(pos, root_node);
  stack[base + tos] = current_node | (b << 24);
  tos += inc;

  while(tos > 0){
    current_node = stack[base + tos -inc] & 0x00ffffff;
    b = (stack[base + tos -inc] & 0xff000000) >> 24;

    if(b > 255){
      return 0;
    }

    unsigned int t = 1;
    while((b & t) == 0){
      t = t << 1;
    }
    current_node += (unsigned int)log2((float)t);
    b = b ^ t;

    if(b == 0){
      tos -= inc;
    }

    else{
      stack[base + tos-inc] = (stack[base + tos -inc] & 0x00ffffff) | (b << 24);
    }

    Node n;
    
    if(current_node < NODE_CACHE_SIZE){
      n = node_cache[current_node];
    }
    else{
      n = nodes[current_node];
    }

    if(n.is_leaf){
      for(short c = 0; c < n.num_children; c++){
#ifdef TEXTURE
        float4 point = tex1Dfetch(pointTexture, (long int)n.pointer + c);
#else
        //float4 p = __ldg((const float4*)(&points[(long int)n.pointer + c]));
        //Point point; point.x = p.x; point.y = p.y; point.z = p.z; point.intensity = p.w;
        Point point = points[(long int)n.pointer + c];
#endif

        real_t dx = pos.x - point.x;
        real_t dy = pos.y - point.y;
        real_t dz = pos.z - point.z;

        real_t distance = sqrt(dx*dx*aniso_x + dy*dy*aniso_y + dz*dz*aniso_z);

        if(distance < INTERPOLATION_RADIUS_D){
#ifdef TEXTURE
          intensity += (1/distance)*point.w;
#else
          intensity += (1/distance)*point.intensity;
#endif
          weight += (1/distance);
        }
      }
    }
    else{
      b = get_covered_subnodes(pos, nodes[current_node]);
      unsigned int temp = (unsigned int)nodes[current_node].pointer;
      stack[base + tos] = (temp) | (b << 24);
      tos += inc;
    }
  }

  if(intensity <= 0){
    return 0;
  }

  real_t ratio = intensity/weight;
  if(ratio <= 1){
    return 0;
  }

  return log(ratio);
}

__device__ real_t get_intensity_for_pos_d(Coord pos, Node* nodes, Point* points){

  Node n = root_node;
  Ranges ranges = top_ranges;
  real_t d = 0;

  while(n.is_leaf == 0){
    d++;
    int index = get_index_d(pos, ranges);
    ranges = get_ranges_for_index_d(ranges, index);

    n = nodes[((long int)n.pointer) + index];
  }

  return interpolate(n, pos, points);
}

__device__ void create_rays(int index, Ray* rays, real_t pixel_width, int res){

  short c = (index % res) - (res/2);
  short d = (index / res) - (res/2);

  Coord start = screen_center_d + up * (d*pixel_width) + right * (c*pixel_width);

  Coord dir = start - eye;

  Coord t1 = (top_d - start)/dir;
  Coord t2 = (bottom_d - start)/dir;

  Coord tmin = fminf(t1, t2);
  Coord tmax = fmaxf(t1, t2);

  real_t tnear = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
  real_t tfar = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

  if(tfar <= tnear){
    rays[index].distance = 0;
    return;
  }

  Coord far = start + dir * tfar;

  tnear = fmax(0, tnear);
  start = start + dir * tnear;

  Coord diff = start - far;

  rays[index].distance = sqrt(diff.x*diff.x + diff.y*diff.y + diff.z*diff.z);

  rays[index].start = start;
  rays[index].dir = dir;

  rays[index].color.r = -1;
  rays[index].color.g = -1;
  rays[index].color.b = -1;
  rays[index].color.a = -1;
}

#ifdef TEXTURE
__global__ void kernel(Node* nodes, Display_color* image, Ray* rays, Color* colors, unsigned int* stack, int stack_size, real_t pixel_width, int offset, int multigpu){
#else
__global__ void kernel(const __restrict__ Point* points, Node* nodes, Display_color* image, Ray* rays, Color* colors, unsigned int* stack, int stack_size, real_t pixel_width, int offset, int multigpu){
#endif

	/*
	int virtualBlockId = blockIdx.x * 10;
	while(virtualBlockId >= gridDim.x){
		virtualBlockId = virtualBlockId - (gridDim.x -1);
	}
	*/


  int i = (blockIdx.x * blockDim.x + threadIdx.x) + offset;
  
  if(multigpu == 0){
    create_rays(i, rays, pixel_width, RESOLUTION_D);
  }

  __shared__ Node node_cache[NODE_CACHE_SIZE];

  if(threadIdx.x < NODE_CACHE_SIZE){
    node_cache[threadIdx.x] = nodes[threadIdx.x];
  }
  __syncthreads();
  

if(rays[i].distance <= 0){
    Display_color b = {0,0,0,0};
    image[i -offset] = b;
    return;
  }

  rays[i] = normalize_ray(rays[i]);

  Coord pos = rays[i].start;
  real_t acc_distance = 0;
  Color output = {0.0,0.0,0.0,0.0};
  
  real_t local_step_size = STEP_SIZE_D;

  while(acc_distance < rays[i].distance){
#ifdef TEXTURE
    real_t intensity = get_intensity_for_pos_full(pos, nodes, node_cache, stack, i*stack_size);
#else
    real_t intensity = get_intensity_for_pos_full(pos, points, nodes, node_cache, stack, i*stack_size);
#endif
    
    if(intensity > 0 && local_step_size > STEP_SIZE_D){
        acc_distance -= local_step_size;
        pos = pos + (rays[i].dir*(-1*local_step_size));
        local_step_size = STEP_SIZE_D;
    }
    else if(intensity == 0 && (local_step_size * STEP_FACTOR_D) <= STEP_LIMIT_D){
        local_step_size *= STEP_FACTOR_D;
    }
    else{

      output = blend_d(output, intensity, colors);

      if(output.a > 0.99f){
        break;
      }
    }

    pos = pos + (rays[i].dir*local_step_size);
    acc_distance += local_step_size; 
  }

  image[i - offset] = to_display_color_d(output);
}


int getPowerfullness(hipDeviceProp_t* p){
	int cudaCores = 0;
	if(p->major == 1){
		cudaCores = 8;
	}
	if(p->major == 2 && p->minor == 0){
		cudaCores = 32;
	}
	if(p->major == 2 && p->minor == 1){
		cudaCores = 48;
	}
	if(p->major == 3){
		cudaCores = 192;
	}

	cudaCores = cudaCores*p->multiProcessorCount;

	return cudaCores*p->clockRate;
}

float logit(float x){
	float y = log(x/(1-x));
	y = y + 5;
	y = y / 10;
	return y;
}

void printError(hipError_t error, char* message){
    if(error != hipSuccess){
        printf("%s\n", hipGetErrorString(error));
        printf("%s\n", message);
    }
}

extern "C" void copy_to_devices_invariant(int deviceCount, Tree* tree, Raycreator* rc){
	for(int device = 0; device < deviceCount; device++){
		hipSetDevice(device);

		hipMalloc((void**)&nodes_d[device], sizeof(Node)*tree->node_list->current_pos);
		hipMalloc((void**)&points_d[device], sizeof(Point)*tree->total_num_points);
		hipMalloc((void**)&image_d[device], sizeof(Display_color)*RESOLUTION*RESOLUTION);
		hipMalloc((void**)&rays_d[device], sizeof(Ray)*RESOLUTION*RESOLUTION);
		hipMalloc((void**)&stack_d[device], sizeof(unsigned int)*RESOLUTION*RESOLUTION*10);//tree->max_depth + 2);
		hipMalloc((void**)&colors_d[device], sizeof(Color)*transfer_overlay->color_table_size);
    printError(hipGetLastError(), "Problem mallocing");

		hipMemcpy(nodes_d[device], tree->node_list->nodes, sizeof(Node)*tree->node_list->current_pos, hipMemcpyHostToDevice);
		hipMemcpy(points_d[device], tree->point_list->points, sizeof(Point)*tree->total_num_points, hipMemcpyHostToDevice);
		hipMemcpy(colors_d[device], transfer_overlay->color_table, sizeof(Color)*transfer_overlay->color_table_size, hipMemcpyHostToDevice);
    printError(hipGetLastError(), "Memcpy");

#ifdef TEXTURE
		hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32,32,32,32, hipChannelFormatKindFloat);
		pointTexture.filterMode = hipFilterModePoint;
		pointTexture.normalized = false;
		hipBindTexture(NULL, pointTexture, points_d[device], channelDesc, sizeof(Point)*tree->total_num_points);
    printError(hipGetLastError(), "texture");
#endif

		hipMemcpyToSymbol(HIP_SYMBOL(top_ranges), &tree->ranges, sizeof(Ranges));
		hipMemcpyToSymbol(HIP_SYMBOL(root_node), &tree->root, sizeof(Node));
		hipMemcpyToSymbol(HIP_SYMBOL(MAX_INTENSITY_D), &MAX_INTENSITY, sizeof(real_t));
		hipMemcpyToSymbol(HIP_SYMBOL(RESOLUTION_D), &RESOLUTION, sizeof(int));
    printError(hipGetLastError(), "memcpy to symbol");
	}
}


void copy_to_devices(int deviceCount, Tree* tree, Raycreator* rc){

	Coord top = {rc->ranges->xmax, rc->ranges->ymax, rc->ranges->zmax};
	Coord bottom = {rc->ranges->xmin, rc->ranges->ymin, rc->ranges->zmin};

    real_t x_radius = INTERPOLATION_RADIUS;
    real_t y_radius = INTERPOLATION_RADIUS;
    real_t z_radius = INTERPOLATION_RADIUS;

    if(ANISOTROPIC){
        x_radius *= (1/sqrt(ANISO_MATRIX[0]));
        y_radius *= (1/sqrt(ANISO_MATRIX[4]));
        z_radius *= (1/sqrt(ANISO_MATRIX[8]));
    }
 		
	for(int device = 0; device < deviceCount; device++){
		hipSetDevice(device);

		hipMemcpy(rays_d[device], rays, sizeof(Ray)*RESOLUTION*RESOLUTION, hipMemcpyHostToDevice);
    printError(hipGetLastError(), "Memcpy");

		hipMemcpyToSymbol(HIP_SYMBOL(STEP_SIZE_D), &STEP_SIZE, sizeof(real_t));
		hipMemcpyToSymbol(HIP_SYMBOL(STEP_FACTOR_D), &STEP_FACTOR, sizeof(real_t));
		hipMemcpyToSymbol(HIP_SYMBOL(STEP_LIMIT_D), &STEP_LIMIT, sizeof(real_t));
		hipMemcpyToSymbol(HIP_SYMBOL(INTERPOLATION_RADIUS_D), &INTERPOLATION_RADIUS, sizeof(real_t));
		hipMemcpyToSymbol(HIP_SYMBOL(X_RADIUS), &x_radius, sizeof(real_t));
		hipMemcpyToSymbol(HIP_SYMBOL(Y_RADIUS), &y_radius, sizeof(real_t));
		hipMemcpyToSymbol(HIP_SYMBOL(Z_RADIUS), &z_radius, sizeof(real_t));
		hipMemcpyToSymbol(HIP_SYMBOL(aniso_x), &ANISO_MATRIX[0], sizeof(real_t));
		hipMemcpyToSymbol(HIP_SYMBOL(aniso_y), &ANISO_MATRIX[4], sizeof(real_t));
		hipMemcpyToSymbol(HIP_SYMBOL(aniso_z), &ANISO_MATRIX[8], sizeof(real_t));
		
		hipMemcpyToSymbol(HIP_SYMBOL(up), &rc->up, sizeof(Coord));
		hipMemcpyToSymbol(HIP_SYMBOL(top_d), &top, sizeof(Coord));
		hipMemcpyToSymbol(HIP_SYMBOL(bottom_d), &bottom, sizeof(Coord));
		hipMemcpyToSymbol(HIP_SYMBOL(eye), &rc->eye, sizeof(Coord));
		hipMemcpyToSymbol(HIP_SYMBOL(right), &rc->right, sizeof(Coord));
		hipMemcpyToSymbol(HIP_SYMBOL(screen_center_d), &rc->screen_center, sizeof(Coord));
		
    printError(hipGetLastError(), "memcpy to symbol");
	}
}

float* get_work_fractions(int deviceCount, int useFiftyFifty){
	float* workFractions = (float*) malloc(sizeof(float) * deviceCount);
	if(stored_work_fractions[0] != 0){
		for (int d = 0; d < deviceCount; ++d) {
			workFractions[d] = stored_work_fractions[d];
		}
	}
	else{
		if(useFiftyFifty != 0){
			workFractions[0] = 0.5;
			workFractions[1] = 0.5;
		}
		float sum = 0;
		for(int d = 0; d < deviceCount; d++){
			hipDeviceProp_t p;
			hipGetDeviceProperties(&p, d);
			workFractions[d] = getPowerfullness(&p);
			sum += workFractions[d];
		}
		for(int d = 0; d < deviceCount; d++){
			workFractions[d] = workFractions[d]/sum;
		}
	}

	return workFractions;
}

void get_work_fractions_logit(float* workFractions){
	workFractions[0] = logit(workFractions[0]);
	workFractions[1] = 1- workFractions[0];
}

void get_work_fractions_ray_length(float* workFractions){

	real_t totalLength = 0;
	for(int i = 0; i < RESOLUTION*RESOLUTION; i++){
		totalLength += rays[i].distance;
	}

	int i = 0;
	real_t partial_length = 0;
	real_t target_partial_length = workFractions[0] * totalLength;
	while(partial_length < target_partial_length){
		partial_length += rays[i].distance;
		i++;
	}

	workFractions[0] = (real_t)i/(real_t)(RESOLUTION*RESOLUTION);
	workFractions[1] = 1 - workFractions[0];
}

int* get_blocks_pr_device(int deviceCount, float* originalWorkFractions, int nBlocks, int useLogit, int useRayLength){
	float workFractions[deviceCount];
	for (int i = 0; i < deviceCount; ++i) {
		workFractions[i] = originalWorkFractions[i];
	}

	if(useLogit != 0){
		get_work_fractions_logit(workFractions);
	}
	if(useRayLength != 0){
		get_work_fractions_ray_length(workFractions);
	}

	int* blocksPrDevice = (int*)malloc(sizeof(int) * deviceCount);
	for (int d = 0; d < deviceCount; ++d) {
			blocksPrDevice[d] = (int)(workFractions[d] * nBlocks);
			printf("Work for device %d: %f, %d/%d\n", d, workFractions[d], blocksPrDevice[d], nBlocks);
	}

	int sum = 0;
	do{
		sum = 0;
		for (int d = 0; d < deviceCount; ++d) {
			sum += blocksPrDevice[d];
		}
		if(sum < nBlocks){
			blocksPrDevice[0]++;
		}
		else if(sum > nBlocks){
			blocksPrDevice[0]--;
		}
	}while(sum != nBlocks);

	return blocksPrDevice;
}


extern "C" void launch_ray_trace_kernel(void* v, Raycreator* rc){

	Tree* tree = (Tree*)v;

	int deviceCount = 0;
    float* workFractions;
    if(MULTIGPU == 1){
        hipGetDeviceCount(&deviceCount);

        int useFiftyFifty = 0;
        workFractions = get_work_fractions(deviceCount, useFiftyFifty);
    }
    else{
        deviceCount = 1;
        workFractions = (float*)malloc(sizeof(float)*1);
        workFractions[0] = 1.0;
    }

	printf("Using %d devices\n", deviceCount);

	copy_to_devices(deviceCount, tree, rc);

	int nThreads=128;
	int totalThreads = RESOLUTION*RESOLUTION;
	int nBlocks = totalThreads/nThreads;
	nBlocks += ((totalThreads%nThreads)>0)?1:0;

    int* blocksPrDevice;
    if(MULTIGPU == 1){
        int useLogit = 0;
        int useRayLength = 1;
        blocksPrDevice = get_blocks_pr_device(deviceCount, workFractions, nBlocks, useLogit, useRayLength);
    }
    else{
        blocksPrDevice = (int*)malloc(sizeof(int)*1);
        blocksPrDevice[0] = nBlocks;
    }

	hipEvent_t startEvents[deviceCount];
	hipEvent_t endEvents[deviceCount];
	hipStream_t streams[deviceCount];

	for(int device = 0; device < deviceCount; device++){
		int start = 0;
		if(device > 0){
			start = blocksPrDevice[device -1] * nThreads;
		}
		hipSetDevice(device);

		hipEventCreate(&startEvents[device]);
		hipEventCreate(&endEvents[device]);
		hipStreamCreate(&streams[device]);
#ifdef TEXTURE
		hipEventRecord(startEvents[device]);
		kernel<<<blocksPrDevice[device], nThreads,0, streams[device]>>>(nodes_d[device], image_d[device], rays_d[device], colors_d[device], stack_d[device], 10, rc->pixel_width, start, MULTIGPU);//tree->max_depth + 2);
		hipEventRecord(endEvents[device]);
#else
		hipEventRecord(startEvents[device]);
		printf("start %d blocks: %d\n", start, blocksPrDevice[device]);
		kernel<<<blocksPrDevice[device], nThreads,0, streams[device]>>>(points_d[device], nodes_d[device], image_d[device], rays_d[device], colors_d[device], stack_d[device], 10, rc->pixel_width, start, MULTIGPU);//tree->max_depth + 2);
		hipEventRecord(endEvents[device]);

#endif
	}

	float r[deviceCount];
	for (int d = 0; d < deviceCount; ++d) {
		hipEventSynchronize(endEvents[d]);
		float time;
		hipEventElapsedTime(&time,startEvents[d], endEvents[d]);
		r[d] = time/workFractions[d];
		printf("Time for device %d: %f\n", d, time);

	}
	float total = r[0] + r[1];
	stored_work_fractions[0] = r[1]/total;
	stored_work_fractions[1] = r[0]/total;


	int previousStart = 0;
	for(int device = 0; device < deviceCount; device++){
		hipSetDevice(device);

		int images_index = log2((real_t)RESOLUTION) - 4;

		int offset = previousStart;
		previousStart = offset + blocksPrDevice[device]*nThreads;

		hipMemcpy(&images[images_index][offset], image_d[device], sizeof(Display_color)*blocksPrDevice[device]*nThreads, hipMemcpyDeviceToHost);
	}

    //freeAndReset(deviceCount);
}

extern "C" void freeAndReset(){
    int deviceCount = 0;
    if(MULTIGPU == 1){
        hipGetDeviceCount(&deviceCount);
    }
    else{
        deviceCount = 1;
    }
    for(int device = 0; device < deviceCount; device++){
        hipSetDevice(device);

        hipFree(nodes_d[device]);
        hipFree(points_d[device]);
        hipFree(image_d[device]);
        hipFree(rays_d[device]);
        hipFree(colors_d[device]);
        hipFree(stack_d[device]);
        hipDeviceReset();
        printError(hipGetLastError(), "Free");
    }
}

extern "C" void initDevice(Tree* tree, Raycreator* rc){
    int deviceCount = 0;
    if(MULTIGPU == 1){
        hipGetDeviceCount(&deviceCount);
    }
    else{
        deviceCount = 1;
    }
    copy_to_devices_invariant(deviceCount, tree, rc);
}

extern "C" long int get_memory_size(){
  hipDeviceProp_t devProp;
  hipGetDeviceProperties(&devProp, 0);
  return devProp.totalGlobalMem;
}
