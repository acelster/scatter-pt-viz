#include "hip/hip_runtime.h"
// Copyright (c) 2013, Thomas L. Falch
// For conditions of distribution and use, see the accompanying LICENSE and README files

// This file is a part of the Scattered Point Visualization application
// developed at the Norwegian University of Science and Technology

#include <hip/hip_runtime.h>
#include <stdio.h>

#include "settings.h"
#include "point.h"
#include "color.h"
#include "grid.h"
#include "global.h"
#include "real.h"
#include "raycreator.h"

namespace g{
__device__ __constant__ real_t STEP_SIZE_D;
__device__ __constant__ real_t INTERPOLATION_RADIUS_D;
__device__ __constant__ Ranges top_ranges;
__device__ __constant__ int x_size;
__device__ __constant__ int y_size;
__device__ __constant__ int z_size;
__device__ __constant__ int num_points;

inline __device__ Coord make_Coord(real_t x, real_t y, real_t z){
  Coord c = {x,y,z};
  return c;
}

inline __device__ Coord operator+(Coord a, Coord b){
  return make_Coord(a.x + b.x, a.y + b.y, a.z + b.z);
}

inline __device__ Coord operator*(Coord a, real_t s){
  return make_Coord(a.x * s, a.y * s, a.z * s);
}

__device__ Ray normalize_ray(Ray r){
  Ray t = r;
  real_t d = sqrt(r.dir.x*r.dir.x + r.dir.y*r.dir.y + r.dir.z*r.dir.z);
  t.dir.x = t.dir.x / d;
  t.dir.y = t.dir.y / d;
  t.dir.z = t.dir.z / d;

  return t;
}

__device__ Color blend_d(Color c, real_t intensity, Color* colors){
  Color new_color, output;

  new_color = colors[(int)((intensity/10.59f)*10000)];

  output.r = c.r + (1-c.a)*new_color.a*new_color.r;
  output.g = c.g + (1-c.a)*new_color.a*new_color.g;
  output.b = c.b + (1-c.a)*new_color.a*new_color.b;

  output.a = c.a + (1-c.a)*new_color.a;

  return output;
}

__device__ Display_color to_display_color_d(Color c){
  Display_color dc;

  if(c.r > 1.0f){
    c.r = 1.0f;
  }

  if(c.b > 1.0f){
    c.b = 1.0f;
  }

  if(c.g > 1.0f){
    c.g = 1.0f;
  }
  
  dc.a = (unsigned char)(c.a*255);
  dc.r = (unsigned char)(c.r*255);
  dc.g = (unsigned char)(c.g*255);
  dc.b = (unsigned char)(c.b*255);

  return dc;
}

__device__ real_t get_intensity_for_pos_d(Coord pos, Grid_cell* indices, Point* points){
  short x = x_size * ((pos.x - top_ranges.xmin)/(top_ranges.xmax - top_ranges.xmin));
  short y = y_size * ((pos.y - top_ranges.ymin)/(top_ranges.ymax - top_ranges.ymin));
  short z = z_size * ((pos.z - top_ranges.zmin)/(top_ranges.zmax - top_ranges.zmin));

  short sub_x = (2*x_size) * ((pos.x - top_ranges.xmin)/(top_ranges.xmax - top_ranges.xmin));
  short sub_y = (2*y_size) * ((pos.y - top_ranges.ymin)/(top_ranges.ymax - top_ranges.ymin));
  short sub_z = (2*z_size) * ((pos.z - top_ranges.zmin)/(top_ranges.zmax - top_ranges.zmin));

  sub_x = sub_x - (x*2);
  sub_y = sub_y - (y*2);
  sub_z = sub_z - (z*2);

  char off[3*8] =  {
    1,1,1,1,0,0,0,0,
    1,1,0,0,1,1,0,0,
    1,0,1,0,1,0,1,0};

  for(char c = 0; c < 8; c ++){
    if(sub_x == 0){
      off[c] *= -1;
    }
    if(sub_y == 0){
      off[8+c] *= -1;
    }
    if(sub_z == 0){
      off[16+c] *= -1;
    }
  }

  real_t intensity = 0;
  real_t weight = 0;

  for(char c = 0; c < 8; c++){
    int index = (z+off[16+c])*x_size*y_size + (y+off[8+c])*x_size + (x+off[c]);

    if(index < 0 || (x+off[c]) >= x_size || (y+off[8+c]) >= y_size || (z+off[16+c]) >= z_size){
      continue;
    }
    if(indices[index].index == -1){
      continue;
    }

    for(int i = indices[index].index; i < indices[index].index + indices[index].length; i++){

      real_t dx = pos.x - points[i].x;
      real_t dy = pos.y - points[i].y;
      real_t dz = pos.z - points[i].z;

      real_t distance = sqrt(dx*dx + dy*dy + dz*dz);

      if(distance < INTERPOLATION_RADIUS_D){
        intensity += (1/distance) * points[i].intensity;
        weight += (1/distance);
      }
    }
  }

  if(intensity <= 0){
    return 0;
  }

  real_t ratio = intensity/weight;
  if(ratio <= 1){
    return 0;
  }

  return log(ratio);
}

__global__ void kernel(Point* points, Grid_cell* indices, Display_color* image, Ray* rays, Color* colors){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
 
  if(rays[i].distance <= 0){
    Display_color b = {0,0,0,0};
    image[i] = b;
    return;
  }

  rays[i] = normalize_ray(rays[i]);

  Coord pos = rays[i].start;
  real_t acc_distance = 0;
  Color output = {0.0,0,0,0.0};

  while(acc_distance < rays[i].distance){
    real_t intensity = get_intensity_for_pos_d(pos, indices, points);

    output = blend_d(output, intensity, colors);

    if(output.a > 0.99f){
      break;
    }

    pos = pos + (rays[i].dir*STEP_SIZE_D);
    acc_distance += STEP_SIZE_D; 
  }

  image[i] = to_display_color_d(output);
}
}

extern "C" void launch_ray_trace_kernel_grid(void* v, Raycreator* rc){
  printf("Tracing rays on GPU using grid datastructure!\n");
  Grid* grid = (Grid*)v;
  int grid_size = grid->x_size*grid->y_size*grid->z_size;

  hipSetDevice(0);

  Point* points_d;
  Grid_cell* indices_d;
  Display_color* image_d;
  Ray* rays_d;
  Color* colors_d;
  hipMalloc((void**)&points_d, sizeof(Point)*grid->num_points);
  hipMalloc((void**)&indices_d, sizeof(Grid_cell)*grid_size);
  hipMalloc((void**)&image_d, sizeof(Display_color)*RESOLUTION*RESOLUTION);
  hipMalloc((void**)&rays_d, sizeof(Ray)*RESOLUTION*RESOLUTION);
  hipMalloc((void**)&colors_d, sizeof(Color)*transfer_overlay->color_table_size);
  printf("%s\n", hipGetErrorString(hipGetLastError()));

  hipMemcpy(points_d, grid->points, sizeof(Point)*grid->num_points, hipMemcpyHostToDevice);
  hipMemcpy(indices_d, grid->indices, sizeof(Grid_cell)*grid_size, hipMemcpyHostToDevice);
  hipMemcpy(rays_d, rays, sizeof(Ray)*RESOLUTION*RESOLUTION, hipMemcpyHostToDevice);
  hipMemcpy(colors_d, transfer_overlay->color_table, sizeof(Color)*transfer_overlay->color_table_size, hipMemcpyHostToDevice);
  printf("%s\n", hipGetErrorString(hipGetLastError()));
  
  hipMemcpyToSymbol(HIP_SYMBOL(g::STEP_SIZE_D), &STEP_SIZE, sizeof(real_t));
  hipMemcpyToSymbol(HIP_SYMBOL(g::INTERPOLATION_RADIUS_D), &INTERPOLATION_RADIUS, sizeof(real_t));
  hipMemcpyToSymbol(HIP_SYMBOL(g::top_ranges), &grid->ranges, sizeof(Ranges));
  hipMemcpyToSymbol(HIP_SYMBOL(g::x_size), &grid->x_size, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(g::y_size), &grid->y_size, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(g::z_size), &grid->z_size, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(g::num_points), &grid->num_points, sizeof(int));
  printf("%s\n", hipGetErrorString(hipGetLastError()));

  int nThreads=128;
  int totalThreads = RESOLUTION*RESOLUTION;
  int nBlocks = totalThreads/nThreads;
  nBlocks += ((totalThreads%nThreads)>0)?1:0;

  g::kernel<<<nBlocks, nThreads>>>(points_d, indices_d, image_d, rays_d, colors_d);
  printf("%s\n", hipGetErrorString(hipGetLastError()));

  int images_index = log2((real_t)RESOLUTION) - 4;
  hipMemcpy(images[images_index], image_d, sizeof(Display_color)*RESOLUTION*RESOLUTION, hipMemcpyDeviceToHost);
  printf("%s\n", hipGetErrorString(hipGetLastError()));

  hipFree(indices_d);
  hipFree(points_d);
  hipFree(image_d);
  hipFree(rays_d);
  hipFree(colors_d);
  printf("%s\n", hipGetErrorString(hipGetLastError()));
}

extern "C" void deviceProperties(){
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  printf("Found %d devices\n", deviceCount);
  for(int d = 0; d < deviceCount; d++){
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, d);
    printf("\n");
    printf("Device: %d\n", d);
    printf("%s\n", devProp.name);
    printf("Compute capability: %d.%d\n", devProp.major, devProp.minor);
    printf("Timeout enabled: %d\n", devProp.kernelExecTimeoutEnabled);
    printf("Global memory: %ld\n", devProp.totalGlobalMem);
    printf("Shared memory pr block: %ld\n", devProp.sharedMemPerBlock);
    printf("Registers pr block: %d\n", devProp.regsPerBlock);
  }
}
    
