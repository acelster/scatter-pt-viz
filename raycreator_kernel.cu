#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "ray.h"
#include "raycreator.h"
#include "settings.h"
#include "point.h"
#include "real.h"

//#include "kernel.h"

__device__ __constant__ Coord up;
__device__ __constant__ Coord eye;
__device__ __constant__ Coord screen_center_d;
__device__ __constant__ Coord right;
__device__ __constant__ Ranges ranges; //DEPRECATED
__device__ __constant__ Coord top_d;
__device__ __constant__ Coord bottom_d;


inline __device__ Coord make_Coord(real_t x, real_t y, real_t z){
  Coord c = {x,y,z};
  return c;
}

inline __device__ Coord operator-(Coord a, Coord b){
  return make_Coord(a.x - b.x, a.y - b.y, a.z - b.z);
}

inline __device__ Coord operator+(Coord a, Coord b){
  return make_Coord(a.x + b.x, a.y + b.y, a.z + b.z);
}

inline __device__ Coord operator*(Coord a, real_t s){
  return make_Coord(a.x * s, a.y * s, a.z * s);
}

inline __device__ Coord operator/(Coord a, Coord b){
  return make_Coord(a.x / b.x, a.y / b.y, a.z / b.z);
}

static __inline__ __device__ Coord fminf(Coord a, Coord b){
  return make_Coord(fminf(a.x,b.x), fminf(a.y,b.y), fminf(a.z,b.z));
}

static __inline__ __device__ Coord fmaxf(Coord a, Coord b){
  return make_Coord(fmaxf(a.x,b.x), fmaxf(a.y,b.y), fmaxf(a.z,b.z));
}

__global__ void kernel(Ray* rays, real_t pixel_width, int res){

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  short c = (index % res) - (res/2);
  short d = (index / res) - (res/2);

  Coord start = screen_center_d + up * (d*pixel_width) + right * (c*pixel_width);

  Coord dir = start - eye;

  Coord t1 = (top_d - start)/dir;
  Coord t2 = (bottom_d - start)/dir;

  Coord tmin = fminf(t1, t2);
  Coord tmax = fmaxf(t1, t2);

  real_t tnear = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
  real_t tfar = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

  if(tfar <= tnear){
    rays[index].distance = 0;
    return;
  }

  Coord far = start + dir * tfar;

  tnear = fmax(0, tnear);
  start = start + dir * tnear;

  Coord diff = start - far;

  rays[index].distance = sqrt(diff.x*diff.x + diff.y*diff.y + diff.z*diff.z);

  rays[index].start = start;
  rays[index].dir = dir;

  rays[index].color.r = -1;
  rays[index].color.g = -1;
  rays[index].color.b = -1;
  rays[index].color.a = -1;
}

extern "C" void launch_kernel(Ray* rays,
    Raycreator* rc,
    Coord screen_center,
    real_t pixel_width){

  hipSetDevice(0);
  printf("%s\n", hipGetErrorString(hipGetLastError()));

  Ray* rays_d;
  hipMalloc((void**)&rays_d, sizeof(Ray) * RESOLUTION * RESOLUTION);
  printf("%s\n", hipGetErrorString(hipGetLastError()));

  int nThreads=128;
  int totalThreads = RESOLUTION*RESOLUTION;
  int nBlocks = totalThreads/nThreads;
  nBlocks += ((totalThreads%nThreads)>0)?1:0;

  Coord top = {rc->ranges->xmax, rc->ranges->ymax, rc->ranges->zmax};
  Coord bottom = {rc->ranges->xmin, rc->ranges->ymin, rc->ranges->zmin};


  hipMemcpyToSymbol(HIP_SYMBOL(up), &rc->up, sizeof(Coord));
  hipMemcpyToSymbol(HIP_SYMBOL(top_d), &top, sizeof(Coord));
  hipMemcpyToSymbol(HIP_SYMBOL(bottom_d), &bottom, sizeof(Coord));
  hipMemcpyToSymbol(HIP_SYMBOL(eye), &rc->eye, sizeof(Coord));
  hipMemcpyToSymbol(HIP_SYMBOL(right), &rc->right, sizeof(Coord));
  hipMemcpyToSymbol(HIP_SYMBOL(screen_center_d), &screen_center, sizeof(Coord));
  printf("%s\n", hipGetErrorString(hipGetLastError()));

  kernel<<< nBlocks, nThreads>>>(rays_d, pixel_width, RESOLUTION);
  printf("%s\n", hipGetErrorString(hipGetLastError()));


  hipMemcpy(rays, rays_d, sizeof(Ray) * RESOLUTION*RESOLUTION, hipMemcpyDeviceToHost);
  printf("%s\n", hipGetErrorString(hipGetLastError()));

  hipFree(rays_d);
}

